#include "hip/hip_runtime.h"
#include "cuda_fonction.cuh"

//Copie de la fonction intersection qu'on peut trouver dans la Sphere mais version device
__device__ float intersection_sphere(const CudaRayon& ray, const SphereData& sphere) {
    CudaVecteur3D oc(ray.origine, sphere.center);

    float a = ray.direction.produitScalaire(ray.direction);
    float b = 2.0f * oc.produitScalaire(ray.direction);
    float c = oc.produitScalaire(oc) - sphere.radius * sphere.radius;

    float discri = b * b - 4.0f * a * c;
    if (discri < 0.0f) return -1.0f; //Je pense que en revoyant les maths les math le discriminant peut être négatif donc normalment ça veut dire pas de solution

    float t1 = (-b + sqrtf(discri)) / (2.0f * a);
    float t2 = (-b - sqrtf(discri)) / (2.0f * a);

    if (t1 >= 0.0f){
        return (t2 >= 0.0f && t2 < t1) ? t2 : t1;
    }
    return -1.0f;
}

//Copie de la fonction intersection qu'on peut trouver dans la Plan mais version device
__device__ float intersection_plan(const CudaRayon& ray, const PlanData& plan) {
    CudaVecteur3D AO(ray.origine, plan.A);  // vecteur de A vers O
    float denom = ray.direction.produitScalaire(plan.normal);

    if (fabs(denom) < 1e-6) {
        return -1.0f; // Presque entièrement parallèle au plan
    } 

    float t = AO.produitScalaire(plan.normal) / denom;
    return (t >= 0.0f) ? t : -1.0f;
}

//Kernel qui permet de calculer notre matrice de <t,num> qui correspond au élément les plus proche qui a une intersection
__global__ void calculate_intersections_kernel(
    CudaRayon* rays, int num_rays,
    SphereData* spheres, int num_spheres,
    PlanData* plans, int num_plans,
    float* t_results, int* object_ids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rays) return;

    CudaRayon ray = rays[idx];
    float proche_t = 1e30f;
    int proche_id = -1;
    int object_counter = 0;

    // Test sur les sphère
    for (int i = 0; i < num_spheres; ++i) {
        float t = intersection_sphere(ray, spheres[i]);
        if (t >= 0.0f && t < proche_t) {
            proche_t = t;
            proche_id = object_counter + i; //offset total des objets
        }
    }
    object_counter += num_spheres;

    // Test sur les plans
    for (int i = 0; i < num_plans; ++i) {
        float t = intersection_plan(ray, plans[i]);
        if (t >= 0.0f && t < proche_t) {
            proche_t = t;
            proche_id = object_counter + i; // offset global
        }
    }

    t_results[idx] = proche_t;
    object_ids[idx] = proche_id;
}

//Launcheur qui transforme nos objet en élément qu'un cuda peut prendre et lance accessoirement un kernel
void launch_calculate_intersections(
    std::vector<Rayon>& rays_host,
    std::vector<Object*>& objects,
    float* t_results_host, int* object_ids_host)
{
    int num_rays = rays_host.size();
    int num_spheres = 0;
    int num_plans = 0;

    // Partie séparation des objet en sphère et plan
    std::vector<Sphere*> spheres_host;
    std::vector<Plan*> plans_host;

    for (auto obj : objects) {
        if (auto* s = dynamic_cast<Sphere*>(obj)) {
            spheres_host.push_back(s);
        } else if (auto* p = dynamic_cast<Plan*>(obj)) {
            plans_host.push_back(p);
        }
    }

    num_spheres = spheres_host.size();
    num_plans = plans_host.size();

    // Partie allocation host et remplissage de nos élément dans notre cuda
    std::vector<CudaRayon> rays_host(num_rays);
    for (int i = 0; i < num_rays; ++i) {
        rays_host[i].origine = CudaPoint3D(
            rays_host[i].origine.getX(),
            rays_host[i].origine.getY(),
            rays_host[i].origine.getZ());
        rays_host[i].direction = CudaVecteur3D(
            rays_host[i].direction.getX(),
            rays_host[i].direction.getY(),
            rays_host[i].direction.getZ());
    }

    std::vector<SphereData> spheres_host(num_spheres);
    for (int i = 0; i < num_spheres; ++i) {
        Point3D c = spheres_host[i]->C;
        spheres_host[i].center = CudaPoint3D(c.getX(), c.getY(), c.getZ());
        spheres_host[i].radius = spheres_host[i]->R;
    }

    std::vector<PlanData> plans_host(num_plans);
    for (int i = 0; i < num_plans; ++i) {
        Point3D A = plans_host[i]->A;
        Vecteur3D N = plans_host[i]->normal;
        plans_host[i].A = CudaPoint3D(A.getX(), A.getY(), A.getZ());
        plans_host[i].normal = CudaVecteur3D(N.getX(), N.getY(), N.getZ());
    }

    //On délcare nos 60000000 de pointeur pour pouvoir balancer sur le cpu tranquille
    CudaRayon* rays_dev;
    SphereData* spheres_dev;
    PlanData* plans_dev;
    float* t_results_dev;
    int* object_ids_dev;

    size_t ray_size = num_rays * sizeof(CudaRayon);
    size_t sphere_size = num_spheres * sizeof(SphereData);
    size_t plan_size = num_plans * sizeof(PlanData);
    size_t result_size = num_rays * sizeof(float);
    size_t id_size = num_rays * sizeof(int);

    // Allocation sur le GPU
    hipMalloc(&rays_dev, ray_size);
    hipMalloc(&spheres_dev, sphere_size);
    hipMalloc(&plans_dev, plan_size);
    hipMalloc(&t_results_dev, result_size);
    hipMalloc(&object_ids_dev, id_size);

    // Transfert CPU → GPU
    hipMemcpy(rays_dev, rays_host.data(), ray_size, hipMemcpyHostToDevice);
    hipMemcpy(spheres_dev, spheres_host.data(), sphere_size, hipMemcpyHostToDevice);
    hipMemcpy(plans_dev, plans_host.data(), plan_size, hipMemcpyHostToDevice);


    // Lancage du Kernel
    int threads = 256;
    int blocks = (num_rays + threads - 1) / threads;

    calculate_intersections_kernel<<<blocks, threads>>>(
        rays_dev, num_rays,
        spheres_dev, num_spheres,
        plans_dev, num_plans,
        t_results_dev, object_ids_dev);


    hipDeviceSynchronize();

    // Copie des résultats GPU → CPU
    hipMemcpy(t_results_host, t_results_dev, result_size, hipMemcpyDeviceToHost);
    hipMemcpy(object_ids_host, object_ids_dev, id_size, hipMemcpyDeviceToHost);

    // On free tout le gpu
    hipFree(rays_dev);
    hipFree(spheres_dev);
    hipFree(plans_dev);
    hipFree(t_results_dev);
    hipFree(object_ids_dev);
}