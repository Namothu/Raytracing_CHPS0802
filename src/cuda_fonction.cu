#include "hip/hip_runtime.h"
#include "Object.h"
#include "Sphere.h"
#include "Plan.h"
#include "Vue.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>


__global__ void calculate_intersections_kernel(
    Rayon* rayons, 
    Object** objets_sphere, 
    int num_spheres, 
    Object** objets_plan, 
    int num_plans,
    float* result_t, 
    int* result_num_object,
    int num_rayons
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_rayons) {
        float t_min = 1e30f;  // Distance maximum initiale
        int object_plus_proche = -1;

        // Calcul de l'intersection pour les sphères
        for (int i = 0; i < num_spheres; i++) {
            float t = objets_sphere[i]->intersection(rayons[idx]);
            if (t >= 0.0f && t < t_min) {
                t_min = t;
                object_plus_proche = i;  // Indice de l'objet sphère
            }
        }

        // Calcul de l'intersection pour les plans
        for (int i = 0; i < num_plans; i++) {
            float t = objets_plan[i]->intersection(rayons[idx]);
            if (t >= 0.0f && t < t_min) {
                t_min = t;
                object_plus_proche = num_spheres + i;  // Indice de l'objet plan
            }
        }

        // Stocker les résultats pour ce rayon
        result_t[idx] = t_min;
        result_num_object[idx] = object_plus_proche;
    }
}


/*

__device__ void calculate_matrice_pixel_kernel_sphere( Object** d_listes_des_objects, Light* d_light, float lumiere_ambiante,
    Rayon* d_matrice_rayon, float* d_matrice_pixel, int width, int height, float distance_max_vision) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Coordonnée x du pixel
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Coordonnée y du pixel

    if (i >= width || j >= height) return;

    int object_plus_proche;
    float t_min;
    float t_tmp;

    object_plus_proche = -1;
    t_min = distance_max_vision;

    // Calcul de l'intersection avec tous les objets
    for (int obj_curr = 0; obj_curr < num_objects; obj_curr++) {
        t_tmp = d_listes_des_objects[obj_curr]->intersection(d_matrice_rayon[i * width + j]);

        if (t_tmp >= 0.0 && t_tmp < t_min) {
            t_min = t_tmp;
            object_plus_proche = obj_curr;
        }
    }

    // Calcul de la couleur si un objet a été trouvé
    if (object_plus_proche >= 0) {
        Point3D P = d_matrice_rayon[i * width + j].point_at_t(t_min);
        Materiel M = d_listes_des_objects[object_plus_proche]->calculerCouleur(P, d_light, lumiere_ambiante);

        // Stockage des valeurs de couleur dans la matrice_pixel
        int pixel_index = (i * width + j) * 3;
        d_matrice_pixel[pixel_index] = M.r;
        d_matrice_pixel[pixel_index + 1] = M.g;
        d_matrice_pixel[pixel_index + 2] = M.b;
    } else {
        // Sinon, on met tout à noir
        int pixel_index = (i * width + j) * 3;
        d_matrice_pixel[pixel_index] = 0.0f;
        d_matrice_pixel[pixel_index + 1] = 0.0f;
        d_matrice_pixel[pixel_index + 2] = 0.0f;
    }
}


__host__ void Vue::calculate_matrice_pixel_gpu(vector<Object*> listes_des_objects, Light * light,float lumiere_ambiante) {
    int width = matrice_rayon.size();
    int height = matrice_rayon[0].size();

    // Allocation de la mémoire sur le GPU
    Object** d_listes_des_objects;
    hipMalloc(&d_listes_des_objects, listes_des_objects.size() * sizeof(Object*));
    hipMemcpy(d_listes_des_objects, listes_des_objects.data(), listes_des_objects.size() * sizeof(Object*), hipMemcpyHostToDevice);

    Light* d_light;
    hipMalloc(&d_light, sizeof(Light));
    hipMemcpy(d_light, light, sizeof(Light), hipMemcpyHostToDevice);

    // Allocation de la mémoire pour la matrice_rayons et matrice_pixel
    Ray* d_matrice_rayon;
    hipMalloc(&d_matrice_rayon, width * height * sizeof(Ray));
    hipMemcpy(d_matrice_rayon, matrice_rayon.data(), width * height * sizeof(Ray), hipMemcpyHostToDevice);

    float* d_matrice_pixel;
    hipMalloc(&d_matrice_pixel, width * height * 3 * sizeof(float));

    // Définir les dimensions de la grille et des blocs
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Lancer le kernel CUDA
    calculate_matrice_pixel_kernel_sphere<<<numBlocks, threadsPerBlock>>>(d_listes_des_objects, d_light, lumiere_ambiante, d_matrice_rayon, d_matrice_pixel, width, height, distance_max_vision);

    // Vérification des erreurs CUDA
    hipDeviceSynchronize();

    // Récupérer les résultats sur le GPU
    hipMemcpy(matrice_pixel.data(), d_matrice_pixel, width * height * 3 * sizeof(float), hipMemcpyDeviceToHost);

    // Libérer la mémoire GPU
    hipFree(d_listes_des_objects);
    hipFree(d_light);
    hipFree(d_matrice_rayon);
    hipFree(d_matrice_pixel);
}