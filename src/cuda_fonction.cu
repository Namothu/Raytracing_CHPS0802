#include "hip/hip_runtime.h"
#include "cuda_fonction.cuh"

// Fonction device d'intersection pour les sphères
__device__ float intersect_sphere(const CudaRayon& ray, const SphereData& sphere) {
    CudaVecteur3D oc(ray.origine, sphere.center);

    float a = ray.direction.produitScalaire(ray.direction);
    float b = 2.0f * oc.produitScalaire(ray.direction);
    float c = oc.produitScalaire(oc) - sphere.radius * sphere.radius;

    float discriminant = b * b - 4.0f * a * c;
    if (discriminant < 0.0f) return -1.0f;

    float t1 = (-b + sqrtf(discriminant)) / (2.0f * a);
    float t2 = (-b - sqrtf(discriminant)) / (2.0f * a);

    if (t1 >= 0.0f) return (t2 >= 0.0f && t2 < t1) ? t2 : t1;
    return -1.0f;
}

// Fonction device d'intersection pour les plans
__device__ float intersect_plan(const CudaRayon& ray, const PlanData& plan) {
    CudaVecteur3D AO(ray.origine, plan.A);  // vecteur de A vers O
    float denom = ray.direction.produitScalaire(plan.normal);

    if (fabs(denom) < 1e-6) return -1.0f;  // parallèle au plan

    float t = AO.produitScalaire(plan.normal) / denom;
    return (t >= 0.0f) ? t : -1.0f;
}

__global__ void calculate_intersections_kernel(
    CudaRayon* rays, int num_rays,
    SphereData* spheres, int num_spheres,
    PlanData* plans, int num_plans,
    float* t_results, int* object_ids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rays) return;

    CudaRayon ray = rays[idx];
    float closest_t = 1e30f;
    int closest_id = -1;
    int object_counter = 0;

    // Test les sphères
    for (int i = 0; i < num_spheres; ++i) {
        float t = intersect_sphere(ray, spheres[i]);
        if (t >= 0.0f && t < closest_t) {
            closest_t = t;
            closest_id = object_counter + i; // offset total des objets
        }
    }
    object_counter += num_spheres;

    // Test les plans
    for (int i = 0; i < num_plans; ++i) {
        float t = intersect_plan(ray, plans[i]);
        if (t >= 0.0f && t < closest_t) {
            closest_t = t;
            closest_id = object_counter + i; // offset global
        }
    }

    t_results[idx] = closest_t;
    object_ids[idx] = closest_id;
}


void launch_calculate_intersections(
    std::vector<Rayon>& rays_cpu,
    std::vector<Object*>& objects,
    float* t_results_host, int* object_ids_host)
{
    int num_rays = rays_cpu.size();
    int num_spheres = 0;
    int num_plans = 0;

    // 🔹 Séparer et compter les objets
    std::vector<Sphere*> spheres_cpu;
    std::vector<Plan*> plans_cpu;

    for (auto obj : objects) {
        if (auto* s = dynamic_cast<Sphere*>(obj)) {
            spheres_cpu.push_back(s);
        } else if (auto* p = dynamic_cast<Plan*>(obj)) {
            plans_cpu.push_back(p);
        }
    }

    num_spheres = spheres_cpu.size();
    num_plans = plans_cpu.size();

    // 🔸 Allouer et remplir les versions CUDA
    std::vector<CudaRayon> rays_host(num_rays);
    for (int i = 0; i < num_rays; ++i) {
        rays_host[i].origine = CudaPoint3D(
            rays_cpu[i].origine.getX(),
            rays_cpu[i].origine.getY(),
            rays_cpu[i].origine.getZ());
        rays_host[i].direction = CudaVecteur3D(
            rays_cpu[i].direction.getX(),
            rays_cpu[i].direction.getY(),
            rays_cpu[i].direction.getZ());
    }

    std::vector<SphereData> spheres_host(num_spheres);
    for (int i = 0; i < num_spheres; ++i) {
        Point3D c = spheres_cpu[i]->C;
        spheres_host[i].center = CudaPoint3D(c.getX(), c.getY(), c.getZ());
        spheres_host[i].radius = spheres_cpu[i]->R;
    }

    

    std::vector<PlanData> plans_host(num_plans);
    for (int i = 0; i < num_plans; ++i) {
        Point3D A = plans_cpu[i]->A;
        Vecteur3D N = plans_cpu[i]->normal;
        plans_host[i].A = CudaPoint3D(A.getX(), A.getY(), A.getZ());
        plans_host[i].normal = CudaVecteur3D(N.getX(), N.getY(), N.getZ());
    }

    // 🔹 Pointeurs device
    CudaRayon* rays_dev;
    SphereData* spheres_dev;
    PlanData* plans_dev;
    float* t_results_dev;
    int* object_ids_dev;

    size_t ray_size = num_rays * sizeof(CudaRayon);
    size_t sphere_size = num_spheres * sizeof(SphereData);
    size_t plan_size = num_plans * sizeof(PlanData);
    size_t result_size = num_rays * sizeof(float);
    size_t id_size = num_rays * sizeof(int);

    printf("Coucou interieur fonction 1\n");

    // 🔸 Allocation sur le GPU
    hipMalloc(&rays_dev, ray_size);
    hipMalloc(&spheres_dev, sphere_size);
    hipMalloc(&plans_dev, plan_size);
    hipMalloc(&t_results_dev, result_size);
    hipMalloc(&object_ids_dev, id_size);

    printf("Coucou interieur fonction 2\n");

    // 🔸 Transfert CPU → GPU
    hipMemcpy(rays_dev, rays_host.data(), ray_size, hipMemcpyHostToDevice);
    hipMemcpy(spheres_dev, spheres_host.data(), sphere_size, hipMemcpyHostToDevice);
    hipMemcpy(plans_dev, plans_host.data(), plan_size, hipMemcpyHostToDevice);

    // 🔹 Lancer le kernel
    int threads = 256;
    int blocks = (num_rays + threads - 1) / threads;

    printf("On lance le calculate");

    calculate_intersections_kernel<<<blocks, threads>>>(
        rays_dev, num_rays,
        spheres_dev, num_spheres,
        plans_dev, num_plans,
        t_results_dev, object_ids_dev);

    hipDeviceSynchronize();

    // 🔸 Copie des résultats GPU → CPU
    hipMemcpy(t_results_host, t_results_dev, result_size, hipMemcpyDeviceToHost);
    hipMemcpy(object_ids_host, object_ids_dev, id_size, hipMemcpyDeviceToHost);

    // 🔹 Libération GPU
    hipFree(rays_dev);
    hipFree(spheres_dev);
    hipFree(plans_dev);
    hipFree(t_results_dev);
    hipFree(object_ids_dev);
}


/*

__device__ void calculate_matrice_pixel_kernel_sphere( Object** d_listes_des_objects, Light* d_light, float lumiere_ambiante,
    Rayon* d_matrice_rayon, float* d_matrice_pixel, int width, int height, float distance_max_vision) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Coordonnée x du pixel
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Coordonnée y du pixel

    if (i >= width || j >= height) return;

    int object_plus_proche;
    float t_min;
    float t_tmp;

    object_plus_proche = -1;
    t_min = distance_max_vision;

    // Calcul de l'intersection avec tous les objets
    for (int obj_curr = 0; obj_curr < num_objects; obj_curr++) {
        t_tmp = d_listes_des_objects[obj_curr]->intersection(d_matrice_rayon[i * width + j]);

        if (t_tmp >= 0.0 && t_tmp < t_min) {
            t_min = t_tmp;
            object_plus_proche = obj_curr;
        }
    }

    // Calcul de la couleur si un objet a été trouvé
    if (object_plus_proche >= 0) {
        Point3D P = d_matrice_rayon[i * width + j].point_at_t(t_min);
        Materiel M = d_listes_des_objects[object_plus_proche]->calculerCouleur(P, d_light, lumiere_ambiante);

        // Stockage des valeurs de couleur dans la matrice_pixel
        int pixel_index = (i * width + j) * 3;
        d_matrice_pixel[pixel_index] = M.r;
        d_matrice_pixel[pixel_index + 1] = M.g;
        d_matrice_pixel[pixel_index + 2] = M.b;
    } else {
        // Sinon, on met tout à noir
        int pixel_index = (i * width + j) * 3;
        d_matrice_pixel[pixel_index] = 0.0f;
        d_matrice_pixel[pixel_index + 1] = 0.0f;
        d_matrice_pixel[pixel_index + 2] = 0.0f;
    }
}


__host__ void Vue::calculate_matrice_pixel_gpu(vector<Object*> listes_des_objects, Light * light,float lumiere_ambiante) {
    int width = matrice_rayon.size();
    int height = matrice_rayon[0].size();

    // Allocation de la mémoire sur le GPU
    Object** d_listes_des_objects;
    hipMalloc(&d_listes_des_objects, listes_des_objects.size() * sizeof(Object*));
    hipMemcpy(d_listes_des_objects, listes_des_objects.data(), listes_des_objects.size() * sizeof(Object*), hipMemcpyHostToDevice);

    Light* d_light;
    hipMalloc(&d_light, sizeof(Light));
    hipMemcpy(d_light, light, sizeof(Light), hipMemcpyHostToDevice);

    // Allocation de la mémoire pour la matrice_rayons et matrice_pixel
    Ray* d_matrice_rayon;
    hipMalloc(&d_matrice_rayon, width * height * sizeof(Ray));
    hipMemcpy(d_matrice_rayon, matrice_rayon.data(), width * height * sizeof(Ray), hipMemcpyHostToDevice);

    float* d_matrice_pixel;
    hipMalloc(&d_matrice_pixel, width * height * 3 * sizeof(float));

    // Définir les dimensions de la grille et des blocs
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Lancer le kernel CUDA
    calculate_matrice_pixel_kernel_sphere<<<numBlocks, threadsPerBlock>>>(d_listes_des_objects, d_light, lumiere_ambiante, d_matrice_rayon, d_matrice_pixel, width, height, distance_max_vision);

    // Vérification des erreurs CUDA
    hipDeviceSynchronize();

    // Récupérer les résultats sur le GPU
    hipMemcpy(matrice_pixel.data(), d_matrice_pixel, width * height * 3 * sizeof(float), hipMemcpyDeviceToHost);

    // Libérer la mémoire GPU
    hipFree(d_listes_des_objects);
    hipFree(d_light);
    hipFree(d_matrice_rayon);
    hipFree(d_matrice_pixel);
}*/