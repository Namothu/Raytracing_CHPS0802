#include "hip/hip_runtime.h"
#include "cuda_fonction.cuh"

struct SphereData {
    Point3D center;
    float radius;
};

struct PlanData {
    Point3D A;
    Vecteur3D normal;
};

// Fonction device d'intersection pour les sphères
__device__ float intersection_sphere(const Rayon& ray, const SphereData& sphere) {
    Vecteur3D oc = Vecteur3D(sphere.center, ray.origine);
    float a = ray.direction.produitScalaire(ray.direction);
    float b = 2.0f * oc.produitScalaire(ray.direction);
    float c = oc.produitScalaire(oc) - (sphere.radius * sphere.radius);
    float discriminant = b * b - 4 * a * c;

    if (discriminant < 0.0f) return -1.0f;

    float t1 = (-b + sqrtf(discriminant)) / (2.0f * a);
    float t2 = (-b - sqrtf(discriminant)) / (2.0f * a);

    if (t1 >= 0.0f) {
        return (t2 >= 0.0f && t2 < t1) ? t2 : t1;
    }

    return -1.0f;
}

// Fonction device d'intersection pour les plans
__device__ float intersection_plan(const Rayon& ray, const PlanData& plan) {
    Vecteur3D AO = Vecteur3D(ray.origine, plan.A);
    float denom = ray.direction.produitScalaire(plan.normal);
    if (fabs(denom) < 1e-6f) return -1.0f;
    return AO.produitScalaire(plan.normal) / denom;
}

__global__ void calculate_intersections_kernel(
    Rayon* rayons,
    SphereData* spheres,
    int num_spheres,
    PlanData* plans,
    int num_plans,
    float* result_t,
    int* result_object_id,
    int num_rayons
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_rayons) return;

    Rayon ray = rayons[idx];
    float t_min = 1e30f;
    int obj_id = -1;

    for (int i = 0; i < num_spheres; i++) {
        float t = intersect_sphere(ray, spheres[i]);
        if (t >= 0.0f && t < t_min) {
            t_min = t;
            obj_id = i; // sphères en premier
        }
    }

    for (int i = 0; i < num_plans; i++) {
        float t = intersect_plan(ray, plans[i]);
        if (t >= 0.0f && t < t_min) {
            t_min = t;
            obj_id = num_spheres + i;
        }
    }

    result_t[idx] = t_min;
    result_object_id[idx] = obj_id;
}

void launch_calculate_intersections(
    Rayon* h_rayons,
    int num_rayons,
    Sphere** h_spheres,
    int num_spheres,
    Plan** h_plans,
    int num_plans,
    float* h_result_t,
    int* h_result_object_id
) {
    // === 1. Convertir vers SphereData et PlanData ===
    std::vector<SphereData> spheres_data(num_spheres);
    for (int i = 0; i < num_spheres; ++i) {
        spheres_data[i].center = h_spheres[i]->C;
        spheres_data[i].radius = h_spheres[i]->R;
    }

    std::vector<PlanData> plans_data(num_plans);
    for (int i = 0; i < num_plans; ++i) {
        plans_data[i].A = h_plans[i]->A;
        plans_data[i].normal = h_plans[i]->normal;
    }

    // === 2. Allocation device ===
    Rayon* d_rayons;
    SphereData* d_spheres;
    PlanData* d_plans;
    float* d_result_t;
    int* d_result_object_id;

    hipMalloc(&d_rayons, num_rayons * sizeof(Rayon));
    hipMalloc(&d_spheres, num_spheres * sizeof(SphereData));
    hipMalloc(&d_plans, num_plans * sizeof(PlanData));
    hipMalloc(&d_result_t, num_rayons * sizeof(float));
    hipMalloc(&d_result_object_id, num_rayons * sizeof(int));

    // === 3. Copier les données vers le GPU ===
    hipMemcpy(d_rayons, h_rayons, num_rayons * sizeof(Rayon), hipMemcpyHostToDevice);
    hipMemcpy(d_spheres, spheres_data.data(), num_spheres * sizeof(SphereData), hipMemcpyHostToDevice);
    hipMemcpy(d_plans, plans_data.data(), num_plans * sizeof(PlanData), hipMemcpyHostToDevice);

    // === 4. Lancer le kernel ===
    int threadsPerBlock = 256;
    int blocks = (num_rayons + threadsPerBlock - 1) / threadsPerBlock;

    calculate_intersections_kernel<<<blocks, threadsPerBlock>>>(
        d_rayons, d_spheres, num_spheres,
        d_plans, num_plans,
        d_result_t, d_result_object_id,
        num_rayons
    );

    hipDeviceSynchronize(); // Attendre la fin du kernel

    // === 5. Copier les résultats vers le host ===
    hipMemcpy(h_result_t, d_result_t, num_rayons * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_result_object_id, d_result_object_id, num_rayons * sizeof(int), hipMemcpyDeviceToHost);

    // === 6. Libération de la mémoire ===
    hipFree(d_rayons);
    hipFree(d_spheres);
    hipFree(d_plans);
    hipFree(d_result_t);
    hipFree(d_result_object_id);
}


/*

__device__ void calculate_matrice_pixel_kernel_sphere( Object** d_listes_des_objects, Light* d_light, float lumiere_ambiante,
    Rayon* d_matrice_rayon, float* d_matrice_pixel, int width, int height, float distance_max_vision) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Coordonnée x du pixel
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Coordonnée y du pixel

    if (i >= width || j >= height) return;

    int object_plus_proche;
    float t_min;
    float t_tmp;

    object_plus_proche = -1;
    t_min = distance_max_vision;

    // Calcul de l'intersection avec tous les objets
    for (int obj_curr = 0; obj_curr < num_objects; obj_curr++) {
        t_tmp = d_listes_des_objects[obj_curr]->intersection(d_matrice_rayon[i * width + j]);

        if (t_tmp >= 0.0 && t_tmp < t_min) {
            t_min = t_tmp;
            object_plus_proche = obj_curr;
        }
    }

    // Calcul de la couleur si un objet a été trouvé
    if (object_plus_proche >= 0) {
        Point3D P = d_matrice_rayon[i * width + j].point_at_t(t_min);
        Materiel M = d_listes_des_objects[object_plus_proche]->calculerCouleur(P, d_light, lumiere_ambiante);

        // Stockage des valeurs de couleur dans la matrice_pixel
        int pixel_index = (i * width + j) * 3;
        d_matrice_pixel[pixel_index] = M.r;
        d_matrice_pixel[pixel_index + 1] = M.g;
        d_matrice_pixel[pixel_index + 2] = M.b;
    } else {
        // Sinon, on met tout à noir
        int pixel_index = (i * width + j) * 3;
        d_matrice_pixel[pixel_index] = 0.0f;
        d_matrice_pixel[pixel_index + 1] = 0.0f;
        d_matrice_pixel[pixel_index + 2] = 0.0f;
    }
}


__host__ void Vue::calculate_matrice_pixel_gpu(vector<Object*> listes_des_objects, Light * light,float lumiere_ambiante) {
    int width = matrice_rayon.size();
    int height = matrice_rayon[0].size();

    // Allocation de la mémoire sur le GPU
    Object** d_listes_des_objects;
    hipMalloc(&d_listes_des_objects, listes_des_objects.size() * sizeof(Object*));
    hipMemcpy(d_listes_des_objects, listes_des_objects.data(), listes_des_objects.size() * sizeof(Object*), hipMemcpyHostToDevice);

    Light* d_light;
    hipMalloc(&d_light, sizeof(Light));
    hipMemcpy(d_light, light, sizeof(Light), hipMemcpyHostToDevice);

    // Allocation de la mémoire pour la matrice_rayons et matrice_pixel
    Ray* d_matrice_rayon;
    hipMalloc(&d_matrice_rayon, width * height * sizeof(Ray));
    hipMemcpy(d_matrice_rayon, matrice_rayon.data(), width * height * sizeof(Ray), hipMemcpyHostToDevice);

    float* d_matrice_pixel;
    hipMalloc(&d_matrice_pixel, width * height * 3 * sizeof(float));

    // Définir les dimensions de la grille et des blocs
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Lancer le kernel CUDA
    calculate_matrice_pixel_kernel_sphere<<<numBlocks, threadsPerBlock>>>(d_listes_des_objects, d_light, lumiere_ambiante, d_matrice_rayon, d_matrice_pixel, width, height, distance_max_vision);

    // Vérification des erreurs CUDA
    hipDeviceSynchronize();

    // Récupérer les résultats sur le GPU
    hipMemcpy(matrice_pixel.data(), d_matrice_pixel, width * height * 3 * sizeof(float), hipMemcpyDeviceToHost);

    // Libérer la mémoire GPU
    hipFree(d_listes_des_objects);
    hipFree(d_light);
    hipFree(d_matrice_rayon);
    hipFree(d_matrice_pixel);
}*/